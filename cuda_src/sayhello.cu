#include "hip/hip_runtime.h"
#include "sayhello.cuh"

__global__ void sayHello(int *a, int *b, int *c)  // __global__ 为CUDA的关键字,表示代码在设备端(GPU端)运行, 可以在CPU端被调用
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x;  i < 1; i += gridDim.x * blockDim.x)
    {
        *c = *a + *b;
        printf("Hello CUDA!%d\n",i);
    }      
}

int callforsayHello()
{
    int c =0, a = 1, b = 2;

    int *dev_c = 0;
    int *dev_a = 0;
    int *dev_b = 0;
    //3.请求CUDA设备的内存（显存），执行CUDA函数
    hipMalloc((void**)&dev_c, sizeof(int));
    hipMalloc((void**)&dev_a, sizeof(int));
    hipMalloc((void**)&dev_b, sizeof(int));

    //4.从主机复制数据到设备上
    hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice);

    int N = 100;
    sayHello <<<(N+9)/10, 10 >>> (dev_a, dev_b, dev_c);  // 函数调用,  <<< >>>中的第一个参数表示块的个数, 第二个参数表示每个线程块中线程的个数

    //5.等待设备所有线程任务执行完毕
    hipDeviceSynchronize();

    //6.数据复制到主机，释放占用空间
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    std::cout<< "&c: " <<&c<<std::endl;
    std::cout<< "dev_c: "<<dev_c<<std::endl;
    std::cout << "c: "<<c<<std::endl;

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    return 0;
}
