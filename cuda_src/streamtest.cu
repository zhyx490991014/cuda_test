#include "hip/hip_runtime.h"
#include "sayhello.cuh"

__global__ void sayHello3(int N, int *a, int *b, int *c)  // __global__ 为CUDA的关键字,表示代码在设备端(GPU端)运行, 可以在CPU端被调用
{
    for(int i = blockIdx.x * blockDim.x + threadIdx.x;  i < N; i += gridDim.x * blockDim.x)
    {
        c[i] = a[i] + b[i];
        // printf("%d %d %d\n",a[i],b[i],c[i]);
        printf("Hello CUDA!%d\n",i);
    }      
}

int callforsayHello3()
{
    int N = 2 << 3; //线程数    //16

    int *c = NULL;
    int *a = NULL;
    int *b = NULL;

    //创建stream
    hipStream_t stream = NULL;
    hipStreamCreate (&stream);

    //CUDA 的统一内存
    hipMallocManaged(&a, N * sizeof(int), hipMemAttachHost);
    hipStreamAttachMemAsync (stream, a);
    hipMallocManaged(&b, N * sizeof(int), hipMemAttachHost);
    hipStreamAttachMemAsync (stream, b);
    hipMallocManaged(&c, N * sizeof(int), hipMemAttachHost);
    hipStreamAttachMemAsync (stream, c);

    // hipMemcpyAsync(inputEC, cloudNew->points.data(), sizeof(float) * 4 * sizeEC, hipMemcpyHostToDevice, stream);
    
    for (int i = 0; i < N; i++) {
        a[i] = i+1;
        b[i] = i*2+1;
    }
    hipStreamSynchronize(stream);

    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    sayHello3 <<<numBlocks, blockSize, 0, stream>>> (N, a, b, c);  // 函数调用,  <<< >>>中的第一个参数表示块的个数, 第二个参数表示每个线程块中线程的个数

    //5.等待设备所有线程任务执行完毕
    hipStreamSynchronize(stream);

    std::cout << "c: " << c <<std::endl;
    for (int i = 0; i < N; i++){
        std::cout << "c["<< i <<"]: "<< c[i] <<std::endl;
    }
    
    hipStreamDestroy(stream);
    hipFree(c);
    hipFree(a);
    hipFree(b);
    return 0;
}
